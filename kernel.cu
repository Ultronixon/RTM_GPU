#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<malloc.h>
#include<stdlib.h>
#include <string.h>
#include<time.h>
#include "phase_correction_ricker_decon.cpp"
#include "LSMOrCon_rec_2D.cpp"
#include "GPU_velocity_real.cpp"
#include "DisToTimeAndTimeToDis1D.cpp"
#include "segy.h"
#include "Resample.cpp"
#include "SGYWrite.cpp"
#include <hip/hip_runtime.h>
#define BLOCKSIZE 16
#define pi 3.1415926535898

__global__ void
	Equal(float *Dlf,float *Drt,float *Dup,float *Ddw,float *DFW0,float *DFW1,int NZ,int NX,int N2,int nfdmax)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	int NX1,NZ1,NX2,NZ2,mod_NX,mod_NZ;
	mod_NX=NX-2*N2;
	mod_NZ=NZ-2*N2;
	NX1=mod_NX+N2;
	NZ1=mod_NZ+N2;
	NX2=mod_NX*nfdmax;
	NZ2=mod_NZ*nfdmax;
	if(x<mod_NZ&&y<nfdmax) 
	{
		Dlf[x*nfdmax+y]=DFW0[(N2+x)*NX+N2-y-1];
		Drt[x*nfdmax+y]=DFW0[(N2+x)*NX+y+NX1];
		Dlf[x*nfdmax+y+NZ2]=DFW1[(N2+x)*NX+N2-y-1];
		Drt[x*nfdmax+y+NZ2]=DFW1[(N2+x)*NX+y+NX1];
	}
	if(x<nfdmax&&y<mod_NX)
	{
		Dup[x*mod_NX+y]=DFW0[(N2-x-1)*NX+y+N2];
		Ddw[x*mod_NX+y]=DFW0[(x+NZ1)*NX+y+N2];
		Dup[x*mod_NX+y+NX2]=DFW1[(N2-x-1)*NX+y+N2];
		Ddw[x*mod_NX+y+NX2]=DFW1[(x+NZ1)*NX+y+N2];
	}
	return;
}
__global__ void
	Add(float *Dv,float *DFW2,float *DFW0,float *DFW1,float *Dc,int *DIndex,float tao2,float h2,float vmin,float dv,int nfdmax,int NZ,int NX,int NXZ,int r_u,int r_x,float wavelet,float hzx2_1)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	int l,x1,x2,y1,y2,M_end,M_top,M_M;
	float w1;
	if(x<NZ&&y<NX)
	{
		M_end=DIndex[(int)((Dv[x*NX+y]-vmin)/dv+1.5)];
		M_top=DIndex[(int)((Dv[x*NX+y]-vmin)/dv+0.5)];					
		M_M=M_end-M_top;
		w1=(1.0+hzx2_1)*Dc[M_top]*DFW1[x*NX+y];
		for(l=1;l<M_M;l++)
		{
			x1=x-l;
			x2=x+l;
			y1=y-l;
			y2=y+l;
			if(x1<0) x1=-x1;
			if(x2>=NZ) x2=2*NZ-2-x2;
			if(y1<0) y1=-y1;
			if(y2>=NX) y2=2*NX-2-y2;

			w1+=Dc[l+M_top]*((DFW1[x1*NX+y]+DFW1[x2*NX+y])*hzx2_1+DFW1[x*NX+y1]+DFW1[x*NX+y2]);
		}

		DFW2[x*NX+y]=2*DFW1[x*NX+y]-DFW0[x*NX+y]+Dv[x*NX+y]*Dv[x*NX+y]*tao2*h2*w1;
		if(x==r_u&&y==r_x)
		{
			DFW2[x*NX+y]+=wavelet;
		}
	}
	return;
}

__global__ void
	Add_Con(float *Dv,float *DFW2,float *DFW0,float *DFW1,float *Dc,float tao2,float h2,int nfdmax,int NZ,int NX,int NXZ,int r_u,int r_x,float wavelet,float hzx2_1)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	int l,x1,x2,y1,y2;
	float w1;
	if(x<NZ&&y<NX)
	{
		w1=(1.0+hzx2_1)*Dc[0]*DFW1[x*NX+y];
		for(l=1;l<=nfdmax;l++)
		{
			x1=x-l;
			x2=x+l;
			y1=y-l;
			y2=y+l;
			if(x1<0) x1=-x1;
			if(x2>=NZ) x2=2*NZ-2-x2;
			if(y1<0) y1=-y1;
			if(y2>=NX) y2=2*NX-2-y2;

			w1+=Dc[l]*((DFW1[x1*NX+y]+DFW1[x2*NX+y])*hzx2_1+DFW1[x*NX+y1]+DFW1[x*NX+y2]);
		}

		DFW2[x*NX+y]=2.0*DFW1[x*NX+y]-DFW0[x*NX+y]+Dv[x*NX+y]*Dv[x*NX+y]*tao2*h2*w1;

		if(x==r_u&&y==r_x)
		{
			DFW2[x*NX+y]+=wavelet;
		}
	}
	return;
}

__global__ void
	Hybrid1(float *DFW1,float *DFW0,float *DFW2,float *DFWb,float *Dlf,float *Drt,float *Dup,float *Ddw,float *Dv,float *Dr_1,float *Dw,int NZ,int NX,int N2,int nfdmax,int k,float taoh,float taoh2)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	
	if(x>0&&x<=N2&&y>=(N2-x+2)&&y<(NX-N2-2+x))
	{
		DFWb[(N2-x)*NX+y]=1/(taoh*Dv[(N2-x)*NX+y]+1)*(taoh*Dv[(N2-x)*NX+y]*(DFW2[(N2-x+1)*NX+y]-DFW0[(N2-x+1)*NX+y]+DFW0[(N2-x)*NX+y])-(-2*DFW1[(N2-x)*NX+y]+DFW0[(N2-x)*NX+y]+DFW2[(N2-x+1)*NX+y]-2*DFW1[(N2-x+1)*NX+y]+DFW0[(N2-x+1)*NX+y])+taoh2*Dv[(N2-x)*NX+y]*Dv[(N2-x)*NX+y]*(DFW2[(N2-x+1)*NX+(y+1)]-2*DFW2[(N2-x+1)*NX+y]+DFW2[(N2-x+1)*NX+(y-1)]+DFW0[(N2-x)*NX+(y+1)]-2*DFW0[(N2-x)*NX+y]+DFW0[(N2-x)*NX+(y-1)]));//�ϱ߽�
	}
	if(y>0&&y<=N2&&x>=(N2-y+2)&&x<(NZ-N2-2+y))
	{
		DFWb[x*NX+(N2-y)]=1/(taoh*Dv[x*NX+(N2-y)]+1)*(taoh*Dv[x*NX+(N2-y)]*(DFW2[x*NX+(N2-y+1)]-DFW0[x*NX+(N2-y+1)]+DFW0[x*NX+(N2-y)])-(-2*DFW1[x*NX+(N2-y)]+DFW0[x*NX+(N2-y)]+DFW2[x*NX+(N2-y+1)]-2*DFW1[x*NX+(N2-y+1)]+DFW0[x*NX+(N2-y+1)])+taoh2*Dv[(N2-y)*NX+x]*Dv[(N2-y)*NX+x]*(DFW2[(x+1)*NX+(N2-y+1)]-2*DFW2[x*NX+(N2-y+1)]+DFW2[(x-1)*NX+(N2-y+1)]+DFW0[(x+1)*NX+(N2-y)]-2*DFW0[x*NX+(N2-y)]+DFW0[(x-1)*NX+(N2-y)]));//��߽�
	}
	if(x>0&&x<=N2&&y>=(N2-x+2)&&y<(NX-N2-2+x))
	{
		DFWb[(NZ-N2+x-1)*NX+y]=1/(taoh*Dv[(NZ-N2+x-1)*NX+y]+1)*(taoh*Dv[(NZ-N2+x-1)*NX+y]*(DFW2[(NZ-N2+x-2)*NX+y]-DFW0[(NZ-N2+x-2)*NX+y]+DFW0[(NZ-N2+x-1)*NX+y])-(-2*DFW1[(NZ-N2+x-1)*NX+y]+DFW0[(NZ-N2+x-1)*NX+y]+DFW2[(NZ-N2+x-2)*NX+y]-2*DFW1[(NZ-N2+x-2)*NX+y]+DFW0[(NZ-N2+x-2)*NX+y])+taoh2*Dv[(N2-x)*NX+y]*Dv[(N2-x)*NX+y]*(DFW2[(NZ-N2+x-2)*NX+(y+1)]-2*DFW2[(NZ-N2+x-2)*NX+y]+DFW2[(NZ-N2+x-2)*NX+(y-1)]+DFW0[(NZ-N2+x-1)*NX+(y+1)]-2*DFW0[(NZ-N2+x-1)*NX+y]+DFW0[(NZ-N2+x-1)*NX+(y-1)]));//�±߽�
	}
	if(y>0&&y<=N2&&x>=(N2-y+2)&&x<(NZ-N2-2+y))
	{
		DFWb[x*NX+NX-N2+y-1]=1/(taoh*Dv[x*NX+NX-N2+y-1]+1)*(taoh*Dv[x*NX+NX-N2+y-1]*(DFW2[x*NX+NX-N2+y-2]-DFW0[x*NX+NX-N2+y-2]+DFW0[x*NX+NX-N2+y-1])-(-2*DFW1[x*NX+NX-N2+y-1]+DFW0[x*NX+NX-N2+y-1]+DFW2[x*NX+NX-N2+y-2]-2*DFW1[x*NX+NX-N2+y-2]+DFW0[x*NX+NX-N2+y-2])+taoh2*Dv[(N2-y)*NX+x]*Dv[(N2-y)*NX+x]*(DFW2[(x+1)*NX+NX-N2+y-2]-2*DFW2[x*NX+NX-N2+y-2]+DFW2[(x-1)*NX+NX-N2+y-2]+DFW0[(x+1)*NX+NX-N2+y-1]-2*DFW0[x*NX+NX-N2+y-1]+DFW0[(x-1)*NX+NX-N2+y-1]));//�ұ߽�
	}
	if(x>0&&x<=N2&&y<NX)
	{
		DFWb[(N2-x+1)*NX+(N2-x)]=1/(2*Dr_1[(N2-x+1)*NX+(N2-x)]+1)*(DFW1[(N2-x+1)*NX+(N2-x)]+Dr_1[(N2-x+1)*NX+(N2-x)]*(DFW2[(N2-x+1)*NX+(N2-x+1)]+DFW2[(N2-x+2)*NX+(N2-x)]));
		DFWb[(N2-x)*NX+(N2-x+1)]=1/(2*Dr_1[(N2-x)*NX+(N2-x+1)]+1)*(DFW1[(N2-x)*NX+(N2-x+1)]+Dr_1[(N2-x)*NX+(N2-x+1)]*(DFW2[(N2-x)*NX+(N2-x+2)]+DFW2[(N2-x+1)*NX+(N2-x+1)]));// ���Ͻ�
		DFWb[(N2-x)*NX+(N2-x)]=1/(2*Dr_1[(N2-x)*NX+(N2-x)]+1)*(DFW1[(N2-x)*NX+(N2-x)]+Dr_1[(N2-x)*NX+(N2-x)]*(DFW2[(N2-x)*NX+(N2-x+1)]+DFW2[(N2-x+1)*NX+(N2-x)]));

		DFWb[(NZ-N2+x-2)*NX+NX-N2+x-1]=1/(2*Dr_1[(NZ-N2+x-2)*NX+NX-N2+x-1]+1)*(DFW1[(NZ-N2+x-2)*NX+NX-N2+x-1]+Dr_1[(NZ-N2+x-2)*NX+NX-N2+x-1]*(DFW2[(NZ-N2+x-2)*NX+NX-N2+x-2]+DFW2[(NZ-N2+x-3)*NX+NX-N2+x-1]));
		DFWb[(NZ-N2+x-1)*NX+NX-N2+x-2]=1/(2*Dr_1[(NZ-N2+x-1)*NX+NX-N2+x-2]+1)*(DFW1[(NZ-N2+x-1)*NX+NX-N2+x-2]+Dr_1[(NZ-N2+x-1)*NX+NX-N2+x-2]*(DFW2[(NZ-N2+x-1)*NX+NX-N2+x-3]+DFW2[(NZ-N2+x-2)*NX+NX-N2+x-2]));//���½�
		DFWb[(NZ-N2+x-1)*NX+NX-N2+x-1]=1/(2*Dr_1[(NZ-N2+x-1)*NX+NX-N2+x-1]+1)*(DFW1[(NZ-N2+x-1)*NX+NX-N2+x-1]+Dr_1[(NZ-N2+x-1)*NX+NX-N2+x-1]*(DFW2[(NZ-N2+x-1)*NX+NX-N2+x-2]+DFW2[(NZ-N2+x-2)*NX+NX-N2+x-1]));

		DFWb[(NZ-N2+x-1)*NX+(N2-x+1)]=1/(2*Dr_1[(NZ-N2+x-1)*NX+(N2-x+1)]+1)*(DFW1[(NZ-N2+x-1)*NX+(N2-x+1)]+Dr_1[(NZ-N2+x-1)*NX+(N2-x+1)]*(DFW2[(NZ-N2+x-2)*NX+(N2-x+1)]+DFW2[(NZ-N2+x-1)*NX+(N2-x+2)]));
		DFWb[(NZ-N2+x-2)*NX+(N2-x)]=1/(2*Dr_1[(NZ-N2+x-2)*NX+(N2-x)]+1)*(DFW1[(NZ-N2+x-2)*NX+(N2-x)]+Dr_1[(NZ-N2+x-2)*NX+(N2-x)]*(DFW2[(NZ-N2+x-3)*NX+(N2-x)]+DFW2[(NZ-N2+x-2)*NX+(N2-x+1)]));//���½�
		DFWb[(NZ-N2+x-1)*NX+(N2-x)]=1/(2*Dr_1[(NZ-N2+x-1)*NX+(N2-x)]+1)*(DFW1[(NZ-N2+x-1)*NX+(N2-x)]+Dr_1[(NZ-N2+x-1)*NX+(N2-x)]*(DFW2[(NZ-N2+x-2)*NX+(N2-x)]+DFW2[(NZ-N2+x-1)*NX+(N2-x+1)]));

		DFWb[(N2-x+1)*NX+NX-N2+x-1]=1/(2*Dr_1[(N2-x+1)*NX+NX-N2+x-1]+1)*(DFW1[(N2-x+1)*NX+NX-N2+x-1]+Dr_1[(N2-x+1)*NX+NX-N2+x-1]*(DFW2[(N2-x+1)*NX+NX-N2+x-2]+DFW2[(N2-x+2)*NX+NX-N2+x-1]));
		DFWb[(N2-x)*NX+NX-N2+x-2]=1/(2*Dr_1[(N2-x)*NX+NX-N2+x-2]+1)*(DFW1[(N2-x)*NX+NX-N2+x-2]+Dr_1[(N2-x)*NX+NX-N2+x-2]*(DFW2[(N2-x)*NX+NX-N2+x-3]+DFW2[(N2-x+1)*NX+NX-N2+x-2]));//���Ͻ�
		DFWb[(N2-x)*NX+NX-N2+x-1]=1/(2*Dr_1[(N2-x)*NX+NX-N2+x-1]+1)*(DFW1[(N2-x)*NX+NX-N2+x-1]+Dr_1[(N2-x)*NX+NX-N2+x-1]*(DFW2[(N2-x)*NX+NX-N2+x-2]+DFW2[(N2-x+1)*NX+NX-N2+x-1]));
	}

	return;
}

__global__ void
	Hybrid2(float *DFW1,float *DFW0,float *DFW2,float *DFWb,float *Dlf,float *Drt,float *Dup,float *Ddw,float *Dv,float *Dr_1,float *Dw,int NZ,int NX,int N2,int nfdmax,int k,float taoh,float taoh2)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;

	if (x>0&&x<=N2&&y>=(N2-x+1)&&y<(NX-N2+x))
	{
		DFW2[(N2-x)*NX+y]=(1-Dw[x])*DFW2[(N2-x)*NX+y]+Dw[x]*DFWb[(N2-x)*NX+y];//�ϱ߽�
	}
	if(y>0&&y<=N2&&x>=(N2-y)&&x<(NZ-N2+y-1))
	{
		DFW2[x*NX+(N2-y)]=(1-Dw[y])*DFW2[x*NX+(N2-y)]+Dw[y]*DFWb[x*NX+(N2-y)];//��߽�
	}
	if(x>0&&x<=N2&&y>=(N2-x)&&y<(NX-N2+x-1))
	{
		DFW2[(NZ-N2+x-1)*NX+y]=(1-Dw[x])*DFW2[(NZ-N2+x-1)*NX+y]+Dw[x]*DFWb[(NZ-N2+x-1)*NX+y];//�±߽�
	}
	if(y>0&&y<=N2&&x>=(N2-y+1)&&x<(NZ-N2+y))
	{
		DFW2[x*NX+NX-N2+y-1]=(1-Dw[y])*DFW2[x*NX+NX-N2+y-1]+Dw[y]*DFWb[x*NX+NX-N2+y-1];//�ұ߽�
	}
	return;
}
__global__ void
	Hybrid3(float *DFW1,float *DFW0,float *DFW2,float *DFWb,float *Dlf,float *Drt,float *Dup,float *Ddw,float *Dv,float *Dr_1,float *Dw,int NZ,int NX,int N2,int nfdmax,int k,float taoh,float taoh2)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	int NX1,NZ1,NX2,NZ2,mod_NX,mod_NZ;
	mod_NX=NX-2*N2;
	mod_NZ=NZ-2*N2;
	NX1=mod_NX+N2;
	NZ1=mod_NZ+N2;
	NX2=mod_NX*nfdmax;
	NZ2=mod_NZ*nfdmax;

	if(x<mod_NZ&&y<nfdmax) 
	{
		Dlf[x*nfdmax+y+k*NZ2]=DFW2[(N2+x)*NX+N2-y-1];
		Drt[x*nfdmax+y+k*NZ2]=DFW2[(N2+x)*NX+y+NX1];
	}
	if(x<nfdmax&&y<mod_NX)
	{
		Dup[x*mod_NX+y+k*NX2]=DFW2[(N2-x-1)*NX+y+N2];
		Ddw[x*mod_NX+y+k*NX2]=DFW2[(x+NZ1)*NX+y+N2];
	}
	return;
}

__global__ void
	Deliver(float *DFW1,float *DFW0,float *DFW2,int NZ,int NX)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	if(x<NZ&&y<NX)
	{
		DFW0[x*NX+y]=DFW1[x*NX+y];
		DFW1[x*NX+y]=DFW2[x*NX+y];
	}
	return;
}
__global__ void
	BKEqual(float *Dlf,float *Drt,float *Dup,float *Ddw,float *DFW1,int NZ,int NX,int N2,int nfdmax,int k)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	int NX1,NZ1,NX2,NZ2,mod_NX,mod_NZ;
	mod_NX=NX-2*N2;
	mod_NZ=NZ-2*N2;
	NX1=mod_NX+N2;
	NZ1=mod_NZ+N2;
	NX2=mod_NX*nfdmax;
	NZ2=mod_NZ*nfdmax;
	if(x<mod_NZ&&y<nfdmax) 
	{
		DFW1[(N2+x)*NX+N2-y-1]=Dlf[x*nfdmax+y+NZ2+k*NZ2];
		DFW1[(N2+x)*NX+y+NX1]=Drt[x*nfdmax+y+NZ2+k*NZ2];
	}
	if(x<nfdmax&&y<mod_NX)
	{
		DFW1[(N2-x-1)*NX+y+N2]=Dup[x*mod_NX+y+NX2+k*NX2];
		DFW1[(x+NZ1)*NX+y+N2]=Ddw[x*mod_NX+y+NX2+k*NX2];
	}
	return;
}
__global__ void
	BKAdd_EFF(float *Dv,float *DFW2,float *DFW0,float *DFW1,float *Dc,int *DIndex,float tao2,float h2,float vmin,float dv,int N2,int NZ,int NX,int NXZ,int r_u,int r_x,float wavelet,float hzx2_1)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	int l,x1,x2,y1,y2,M_end,M_top,M_M,NX1,NZ1;
	float w1;
	NX1=NX-N2;
	NZ1=NZ-N2;
	if(x>=N2&&x<NZ1&&y>=N2&&y<NX1)
	{
		M_end=DIndex[(int)((Dv[x*NX+y]-vmin)/dv+1.5)];
		M_top=DIndex[(int)((Dv[x*NX+y]-vmin)/dv+0.5)];					
		M_M=M_end-M_top;
		w1=(1.0+hzx2_1)*Dc[M_top]*DFW1[x*NX+y];
		for(l=1;l<M_M;l++)
		{
			x1=x-l;
			x2=x+l;
			y1=y-l;
			y2=y+l;
			if(x1<0) x1=-x1;
			if(x2>=NZ) x2=2*NZ-2-x2;
			if(y1<0) y1=-y1;
			if(y2>=NX) y2=2*NX-2-y2;

			w1+=Dc[l+M_top]*((DFW1[x1*NX+y]+DFW1[x2*NX+y])*hzx2_1+DFW1[x*NX+y1]+DFW1[x*NX+y2]);
		}

		DFW2[x*NX+y]=2.0*DFW1[x*NX+y]-DFW0[x*NX+y]+Dv[x*NX+y]*Dv[x*NX+y]*tao2*h2*w1;

		if(x==r_u&&y==r_x)
		{
			DFW2[x*NX+y]+=wavelet;
		}
	}
	return;
}


__global__ void
	BKAdd_EFF_Con(float *Dv,float *DFW2,float *DFW0,float *DFW1,float *Dc,float tao2,float h2,int N2,int NZ,int NX,int NXZ,int r_u,int r_x,int nfdmax,float wavelet,float hzx2_1)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	int l,x1,x2,y1,y2,NX1,NZ1;
	float w1;
	NX1=NX-N2;
	NZ1=NZ-N2;
	if(x>=N2&&x<NZ1&&y>=N2&&y<NX1)
	{
		w1=(1.0+hzx2_1)*Dc[0]*DFW1[x*NX+y];
		for(l=1;l<=nfdmax;l++)
		{
			x1=x-l;
			x2=x+l;
			y1=y-l;
			y2=y+l;
			if(x1<0) x1=-x1;
			if(x2>=NZ) x2=2*NZ-2-x2;
			if(y1<0) y1=-y1;
			if(y2>=NX) y2=2*NX-2-y2;

			w1+=Dc[l]*((DFW1[x1*NX+y]+DFW1[x2*NX+y])*hzx2_1+DFW1[x*NX+y1]+DFW1[x*NX+y2]);
		}

		DFW2[x*NX+y]=2.0*DFW1[x*NX+y]-DFW0[x*NX+y]+Dv[x*NX+y]*Dv[x*NX+y]*tao2*h2*w1;

		if(x==r_u&&y==r_x)
		{
			DFW2[x*NX+y]+=wavelet;
		}
	}
	return;
}


__global__ void
	Deliver_EFF(float *DFW1,float *DFW0,float *DFW2,int N2,int NZ,int NX)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	int NX1,NZ1;
	NX1=NX-N2;
	NZ1=NZ-N2;
	if(x>=N2&&x<NZ1&&y>=N2&&y<NX1)
	{
		DFW0[x*NX+y]=DFW1[x*NX+y];
		DFW1[x*NX+y]=DFW2[x*NX+y];
	}
	return;
}

__global__ void
	BKAdd(float *Dv,float *DBW2,float *DBW1,float *DBW0,float *Dc,int *DIndex,float *Dseis,float tao2,float h2,float vmin,float dv,int nfdmax, int NZ,int NX,int NXZ,int s_l,int s_r,int s_z,int ds,float hzx2_1)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	int l,judge,s,x1,x2,y1,y2,M_end,M_top,M_M;
	float w1;

	if(x<NZ&&y<NX)
	{		
		judge=(int)((y-s_l)%ds);
		s=(int)((y-s_l)/ds);
		if(x==s_z&&y>=s_l&&y<=s_r&&judge==0&&Dseis[s]!=0)
		{	
			DBW2[x*NX+y]=Dseis[s];
		}
		else
		{	
			M_end=DIndex[(int)((Dv[x*NX+y]-vmin)/dv+1.5)];
			M_top=DIndex[(int)((Dv[x*NX+y]-vmin)/dv+0.5)];					
			M_M=M_end-M_top;
			w1=(1.0+hzx2_1)*Dc[M_top]*DBW1[x*NX+y];
			for(l=1;l<M_M;l++)
			{
				x1=x-l;
				x2=x+l;
				y1=y-l;
				y2=y+l;
				if(x1<0) x1=-x1;
				if(x2>=NZ) x2=2*NZ-2-x2;
				if(y1<0) y1=-y1;
				if(y2>=NX) y2=2*NX-2-y2;

				w1=w1+Dc[l+M_top]*((DBW1[x1*NX+y]+DBW1[x2*NX+y])*hzx2_1+DBW1[x*NX+y1]+DBW1[x*NX+y2]);
			}

			DBW2[x*NX+y]=2*DBW1[x*NX+y]-DBW0[x*NX+y]+Dv[x*NX+y]*Dv[x*NX+y]*tao2*h2*w1;
		}

	}
	return;
}
__global__ void
	BKAdd_Con(float *Dv,float *DBW2,float *DBW1,float *DBW0,float *Dc,float *Dseis,float tao2,float h2,int nfdmax,int NZ,int NX,int NXZ,int s_l,int s_r,int s_z,int ds,float hzx2_1)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;
	int l,judge,s,x1,x2,y1,y2;
	float w1;

	if(x<NZ&&y<NX)
	{		
		judge=(int)((y-s_l)%ds);
		s=(int)((y-s_l)/ds);
		if(x==s_z&&y>=s_l&&y<=s_r&&judge==0&&Dseis[s]!=0)
		{			
			DBW2[x*NX+y]=Dseis[s];
		}
		else
		{	
			w1=(1.0+hzx2_1)*Dc[0]*DBW1[x*NX+y];
			for(l=1;l<=nfdmax;l++)
			{
				x1=x-l;
				x2=x+l;
				y1=y-l;
				y2=y+l;
				if(x1<0) x1=-x1;
				if(x2>=NZ) x2=2*NZ-2-x2;
				if(y1<0) y1=-y1;
				if(y2>=NX) y2=2*NX-2-y2;

				w1=w1+Dc[l]*((DBW1[x1*NX+y]+DBW1[x2*NX+y])*hzx2_1+DBW1[x*NX+y1]+DBW1[x*NX+y2]);
			}
			DBW2[x*NX+y]=2*DBW1[x*NX+y]-DBW0[x*NX+y]+Dv[x*NX+y]*Dv[x*NX+y]*tao2*h2*w1;
		}

	}
	return;
}


__global__ void
	BKHybrid1(float *DFW1,float *DFW0,float *DFW2,float *DFWb,float *Dv,float *Dr_1,float *Dw,int NZ,int NX,int N2,float taoh,float taoh2)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;

	if(x>0&&x<=N2&&y>=(N2-x+2)&&y<(NX-N2-2+x))
	{
		DFWb[(N2-x)*NX+y]=1/(taoh*Dv[(N2-x)*NX+y]+1)*(taoh*Dv[(N2-x)*NX+y]*(DFW2[(N2-x+1)*NX+y]-DFW0[(N2-x+1)*NX+y]+DFW0[(N2-x)*NX+y])-(-2*DFW1[(N2-x)*NX+y]+DFW0[(N2-x)*NX+y]+DFW2[(N2-x+1)*NX+y]-2*DFW1[(N2-x+1)*NX+y]+DFW0[(N2-x+1)*NX+y])+taoh2*Dv[(N2-x)*NX+y]*Dv[(N2-x)*NX+y]*(DFW2[(N2-x+1)*NX+(y+1)]-2*DFW2[(N2-x+1)*NX+y]+DFW2[(N2-x+1)*NX+(y-1)]+DFW0[(N2-x)*NX+(y+1)]-2*DFW0[(N2-x)*NX+y]+DFW0[(N2-x)*NX+(y-1)]));//�ϱ߽�
	}
	if(y>0&&y<=N2&&x>=(N2-y+2)&&x<(NZ-N2-2+y))
	{
		DFWb[x*NX+(N2-y)]=1/(taoh*Dv[x*NX+(N2-y)]+1)*(taoh*Dv[x*NX+(N2-y)]*(DFW2[x*NX+(N2-y+1)]-DFW0[x*NX+(N2-y+1)]+DFW0[x*NX+(N2-y)])-(-2*DFW1[x*NX+(N2-y)]+DFW0[x*NX+(N2-y)]+DFW2[x*NX+(N2-y+1)]-2*DFW1[x*NX+(N2-y+1)]+DFW0[x*NX+(N2-y+1)])+taoh2*Dv[(N2-y)*NX+x]*Dv[(N2-y)*NX+x]*(DFW2[(x+1)*NX+(N2-y+1)]-2*DFW2[x*NX+(N2-y+1)]+DFW2[(x-1)*NX+(N2-y+1)]+DFW0[(x+1)*NX+(N2-y)]-2*DFW0[x*NX+(N2-y)]+DFW0[(x-1)*NX+(N2-y)]));//��߽�
	}
	if(x>0&&x<=N2&&y>=(N2-x+2)&&y<(NX-N2-2+x))
	{
		DFWb[(NZ-N2+x-1)*NX+y]=1/(taoh*Dv[(NZ-N2+x-1)*NX+y]+1)*(taoh*Dv[(NZ-N2+x-1)*NX+y]*(DFW2[(NZ-N2+x-2)*NX+y]-DFW0[(NZ-N2+x-2)*NX+y]+DFW0[(NZ-N2+x-1)*NX+y])-(-2*DFW1[(NZ-N2+x-1)*NX+y]+DFW0[(NZ-N2+x-1)*NX+y]+DFW2[(NZ-N2+x-2)*NX+y]-2*DFW1[(NZ-N2+x-2)*NX+y]+DFW0[(NZ-N2+x-2)*NX+y])+taoh2*Dv[(N2-x)*NX+y]*Dv[(N2-x)*NX+y]*(DFW2[(NZ-N2+x-2)*NX+(y+1)]-2*DFW2[(NZ-N2+x-2)*NX+y]+DFW2[(NZ-N2+x-2)*NX+(y-1)]+DFW0[(NZ-N2+x-1)*NX+(y+1)]-2*DFW0[(NZ-N2+x-1)*NX+y]+DFW0[(NZ-N2+x-1)*NX+(y-1)]));//�±߽�
	}
	if(y>0&&y<=N2&&x>=(N2-y+2)&&x<(NZ-N2-2+y))
	{
		DFWb[x*NX+NX-N2+y-1]=1/(taoh*Dv[x*NX+NX-N2+y-1]+1)*(taoh*Dv[x*NX+NX-N2+y-1]*(DFW2[x*NX+NX-N2+y-2]-DFW0[x*NX+NX-N2+y-2]+DFW0[x*NX+NX-N2+y-1])-(-2*DFW1[x*NX+NX-N2+y-1]+DFW0[x*NX+NX-N2+y-1]+DFW2[x*NX+NX-N2+y-2]-2*DFW1[x*NX+NX-N2+y-2]+DFW0[x*NX+NX-N2+y-2])+taoh2*Dv[(N2-y)*NX+x]*Dv[(N2-y)*NX+x]*(DFW2[(x+1)*NX+NX-N2+y-2]-2*DFW2[x*NX+NX-N2+y-2]+DFW2[(x-1)*NX+NX-N2+y-2]+DFW0[(x+1)*NX+NX-N2+y-1]-2*DFW0[x*NX+NX-N2+y-1]+DFW0[(x-1)*NX+NX-N2+y-1]));//�ұ߽�
	}
	if(x>0&&x<=N2&&y<NX)
	{
		DFWb[(N2-x+1)*NX+(N2-x)]=1/(2*Dr_1[(N2-x+1)*NX+(N2-x)]+1)*(DFW1[(N2-x+1)*NX+(N2-x)]+Dr_1[(N2-x+1)*NX+(N2-x)]*(DFW2[(N2-x+1)*NX+(N2-x+1)]+DFW2[(N2-x+2)*NX+(N2-x)]));
		DFWb[(N2-x)*NX+(N2-x+1)]=1/(2*Dr_1[(N2-x)*NX+(N2-x+1)]+1)*(DFW1[(N2-x)*NX+(N2-x+1)]+Dr_1[(N2-x)*NX+(N2-x+1)]*(DFW2[(N2-x)*NX+(N2-x+2)]+DFW2[(N2-x+1)*NX+(N2-x+1)]));// ���Ͻ�
		DFWb[(N2-x)*NX+(N2-x)]=1/(2*Dr_1[(N2-x)*NX+(N2-x)]+1)*(DFW1[(N2-x)*NX+(N2-x)]+Dr_1[(N2-x)*NX+(N2-x)]*(DFW2[(N2-x)*NX+(N2-x+1)]+DFW2[(N2-x+1)*NX+(N2-x)]));

		DFWb[(NZ-N2+x-2)*NX+NX-N2+x-1]=1/(2*Dr_1[(NZ-N2+x-2)*NX+NX-N2+x-1]+1)*(DFW1[(NZ-N2+x-2)*NX+NX-N2+x-1]+Dr_1[(NZ-N2+x-2)*NX+NX-N2+x-1]*(DFW2[(NZ-N2+x-2)*NX+NX-N2+x-2]+DFW2[(NZ-N2+x-3)*NX+NX-N2+x-1]));
		DFWb[(NZ-N2+x-1)*NX+NX-N2+x-2]=1/(2*Dr_1[(NZ-N2+x-1)*NX+NX-N2+x-2]+1)*(DFW1[(NZ-N2+x-1)*NX+NX-N2+x-2]+Dr_1[(NZ-N2+x-1)*NX+NX-N2+x-2]*(DFW2[(NZ-N2+x-1)*NX+NX-N2+x-3]+DFW2[(NZ-N2+x-2)*NX+NX-N2+x-2]));//���½�
		DFWb[(NZ-N2+x-1)*NX+NX-N2+x-1]=1/(2*Dr_1[(NZ-N2+x-1)*NX+NX-N2+x-1]+1)*(DFW1[(NZ-N2+x-1)*NX+NX-N2+x-1]+Dr_1[(NZ-N2+x-1)*NX+NX-N2+x-1]*(DFW2[(NZ-N2+x-1)*NX+NX-N2+x-2]+DFW2[(NZ-N2+x-2)*NX+NX-N2+x-1]));

		DFWb[(NZ-N2+x-1)*NX+(N2-x+1)]=1/(2*Dr_1[(NZ-N2+x-1)*NX+(N2-x+1)]+1)*(DFW1[(NZ-N2+x-1)*NX+(N2-x+1)]+Dr_1[(NZ-N2+x-1)*NX+(N2-x+1)]*(DFW2[(NZ-N2+x-2)*NX+(N2-x+1)]+DFW2[(NZ-N2+x-1)*NX+(N2-x+2)]));
		DFWb[(NZ-N2+x-2)*NX+(N2-x)]=1/(2*Dr_1[(NZ-N2+x-2)*NX+(N2-x)]+1)*(DFW1[(NZ-N2+x-2)*NX+(N2-x)]+Dr_1[(NZ-N2+x-2)*NX+(N2-x)]*(DFW2[(NZ-N2+x-3)*NX+(N2-x)]+DFW2[(NZ-N2+x-2)*NX+(N2-x+1)]));//���½�
		DFWb[(NZ-N2+x-1)*NX+(N2-x)]=1/(2*Dr_1[(NZ-N2+x-1)*NX+(N2-x)]+1)*(DFW1[(NZ-N2+x-1)*NX+(N2-x)]+Dr_1[(NZ-N2+x-1)*NX+(N2-x)]*(DFW2[(NZ-N2+x-2)*NX+(N2-x)]+DFW2[(NZ-N2+x-1)*NX+(N2-x+1)]));

		DFWb[(N2-x+1)*NX+NX-N2+x-1]=1/(2*Dr_1[(N2-x+1)*NX+NX-N2+x-1]+1)*(DFW1[(N2-x+1)*NX+NX-N2+x-1]+Dr_1[(N2-x+1)*NX+NX-N2+x-1]*(DFW2[(N2-x+1)*NX+NX-N2+x-2]+DFW2[(N2-x+2)*NX+NX-N2+x-1]));
		DFWb[(N2-x)*NX+NX-N2+x-2]=1/(2*Dr_1[(N2-x)*NX+NX-N2+x-2]+1)*(DFW1[(N2-x)*NX+NX-N2+x-2]+Dr_1[(N2-x)*NX+NX-N2+x-2]*(DFW2[(N2-x)*NX+NX-N2+x-3]+DFW2[(N2-x+1)*NX+NX-N2+x-2]));//���Ͻ�
		DFWb[(N2-x)*NX+NX-N2+x-1]=1/(2*Dr_1[(N2-x)*NX+NX-N2+x-1]+1)*(DFW1[(N2-x)*NX+NX-N2+x-1]+Dr_1[(N2-x)*NX+NX-N2+x-1]*(DFW2[(N2-x)*NX+NX-N2+x-2]+DFW2[(N2-x+1)*NX+NX-N2+x-1]));
	}

	return;
}
__global__ void
	BKHybrid2(float *DFW1,float *DFW0,float *DFW2,float *DFWb,float *Dv,float *Dr_1,float *Dw,int NZ,int NX,int N2,float taoh,float taoh2)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;

	if(x>0&&x<=N2&&y>=(N2-x+1)&&y<(NX-N2+x))
	{
		DFW2[(N2-x)*NX+y]=(1-Dw[x])*DFW2[(N2-x)*NX+y]+Dw[x]*DFWb[(N2-x)*NX+y];//�ϱ߽�
	}
	if(y>0&&y<=N2&&x>=(N2-y)&&x<(NZ-N2+y-1))
	{
		DFW2[x*NX+(N2-y)]=(1-Dw[y])*DFW2[x*NX+(N2-y)]+Dw[y]*DFWb[x*NX+(N2-y)];//��߽�
	}
	if(x>0&&x<=N2&&y>=(N2-x)&&y<(NX-N2+x-1))
	{
		DFW2[(NZ-N2+x-1)*NX+y]=(1-Dw[x])*DFW2[(NZ-N2+x-1)*NX+y]+Dw[x]*DFWb[(NZ-N2+x-1)*NX+y];//�±߽�
	}
	if(y>0&&y<=N2&&x>=(N2-y+1)&&x<(NZ-N2+y))
	{
		DFW2[x*NX+NX-N2+y-1]=(1-Dw[y])*DFW2[x*NX+NX-N2+y-1]+Dw[y]*DFWb[x*NX+NX-N2+y-1];//�ұ߽�
	}
	return;
}

__global__ void
	Rel_NonCompen(float *DBW2,float *DFW2,float *Drel1,float *Drel2,int NZ,int NX)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;

	if(x<NZ&&y<NX)
	{
		Drel1[x*NX+y]+=DBW2[x*NX+y]*DFW2[x*NX+y];
		Drel2[x*NX+y]+=DFW2[x*NX+y]*DFW2[x*NX+y];
	}
	return;
}

__global__ void
	Rel_Compen(float *DBW2,float *DFW2,float *DBWADD,float *DFWADD,float *Drel1,float *Drel2,int NZ,int NX)
{
	int x=blockIdx.x*BLOCKSIZE+threadIdx.x;
	int y=blockIdx.y*BLOCKSIZE+threadIdx.y;

	if(x<NZ&&y<NX)
	{
		DFWADD[x*NX+y]+=DFW2[x*NX+y];
		DBWADD[x*NX+y]+=DBW2[x*NX+y];
		Drel1[x*NX+y]+=DBWADD[x*NX+y]*DFWADD[x*NX+y];
		Drel2[x*NX+y]+=DFW2[x*NX+y]*DFW2[x*NX+y];
	}
	return;
}

float f(float t1,float f0);  //ricker source

void velocity(char *OutNameVp,float *v,float *v_2,float *r,float *r_1,float *r_2,int NZ,int NX,int NXZ,int N2,float tao,float h,int choice) ;

void WriteSGY(float *Data,int NX,int NT,int tao3,float *SX,float *SY,float RX,float RY,float *DSR,char *FILENAME);

int main()
{
	hipSetDevice(0);
	clock_t start, finish;
	float duration;
	start = clock (); 

	char name[100],name1[100],name3[100],OutNameseis[100],OutNameVp[100],OutNameDPR[100],OutPara[100],Result[100];
	int i,j,i1,i2,j1,j2,m,k,N,NC,nvel,mod_NZ,mod_NX,NZ,NX,Nmax,NT2,NT1,NT,NXZ,s_l,s_r,s_z,r_x,r_u,n,nfdmax,nfdmin,N2,nrec,dr,ds;
	int iNorm,iCompen,Nsmooth,iLSTE,nthita,ifv,size,sizec,sizew,sizeseis,sizein,sizeLF,sizeUP,NX_BG,NX_ED,NZ_BG,NZ_ED;
	int *M,*Index,*fdcoeneed,*DIndex;
	float h,hz,tao1,tao,taoh,tao2,h2,taoh2,f0,fmax,eps,df,vmax,vmin,dv,vel,wavelet,MIGmax,whitecoe,stable,hzx,hzx2_1,angle,wthite_phase;
	float *INRE,*FW2,*FW1,*FW0,*FWb,*BW2,*BW1,*BW0,*v,*v_2,*r,*r_1,*r_2,*c,**seis1,**seis,*Cseis,*w,*MIG1,*MIG2,*MIG3;
	float *DFWADD,*DFW2,*DFW0,*DFW1,*DFWb,*DBWADD,*DBW2,*DBW0,*DBW1,*Dv,*Dr,*Dr_1,*Dw,*Dc,*Drel1,*Drel2,*Dseis,*Dlf,*Drt,*Dup,*Ddw;

	FILE *inp,*out,*out1,*DepRec,*Rdseismic,*Af_la;

	inp=fopen("2D_Real_RVSP_RTM.txt","r");

	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&nfdmax);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&nfdmin);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&N2);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%f\n",&f0);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%f\n",&fmax);	
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%f\n",&df);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&nthita);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%f\n",&eps);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%f\n",&dv);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&iLSTE);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&ifv);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%f\n",&whitecoe);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%f\n",&hz);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%f\n",&tao);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&iNorm);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&iCompen);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&Nsmooth);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%f\n",&wthite_phase);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%f\n",&angle);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&NX_BG);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&NX_ED);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&NZ_BG);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%d\n",&NZ_ED);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%s\n",OutNameseis);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%s\n",OutNameVp);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%s\n",OutNameDPR);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%s\n",OutPara);
	fscanf(inp,"%[^\n]\n",name);
	fscanf(inp,"%s\n",Result);
	fclose(inp);

	inp=fopen(OutPara,"r");
	fscanf(inp,"%f \n%f \n%d \n%d \n%d \n%d \n%d \n%d \n%d \n%d \n%d \n%d",&h,&tao1,&mod_NZ,&mod_NX,&NT1,&s_l,&s_z,&n,&ds,&r_x,&nrec,&dr);
	fclose(inp);
	

	s_l=s_l+N2-1;
	r_x=r_x+N2-1;
	s_z=s_z+N2-1;
	s_r=(n-1)*ds+s_l;
	NZ=mod_NZ+2*N2;
	NX=mod_NX+2*N2;
	NT=(int)((NT1-1)*tao1/tao+1.5);
	taoh=tao/h;
	tao2=pow(tao,2);
	h2=1/pow(h,2);
	taoh2=tao2*h2/2;
	NXZ=NZ*NX;
	sizeLF=sizeof (float) *(nfdmax*mod_NZ*NT);
	sizeUP=sizeof (float) *(nfdmax*mod_NX*NT);
	size=sizeof (float) *(NXZ);
	sizew=sizeof(float)*(N2+1);
	sizeseis=sizeof(float)*(n);
	NT2=int(2.0/(f0*tao))+1;	
	hzx=hz/h;
	hzx2_1=1/(hzx*hzx);
	Nmax=NX;
	if(Nmax<NZ) Nmax=NZ;
	printf("ifv=%d\n",ifv);
	printf("Nmax=%d\n",Nmax);
	printf("hzx=%f,hzx2_1=%f\n",hzx,hzx2_1);
	printf("The maximum length of operator\nnfdmax=%d\n",nfdmax);
	printf("The minimum length of operator\nnfdmin=%d\n",nfdmin);
	printf("The hyrid absorbing boundary width\nN2=%d\n",N2);
	printf("space interval\nh=%f\n",h);
	printf("time interval\ntao=%f\n",tao);
	printf("z grid dimension\nmod_NZ=%d\n",mod_NZ);
	printf("x grid dimension\nmod_NX=%d\n",mod_NX);
	printf("actual grid number in z\nNZ=%d\n",NZ);
	printf("actual grid number in x\nNX=%d\n",NX);
	printf("Number of time number\nNT=%d\n",NT);
	printf("Source X\ns_x=%d\n",s_l);
	printf("Source Z\ns_z=%d\n",s_z);	
	printf("The number of sources\nn=%d\n",n);
	printf("The interval of sources\nds=%d\n",ds);
	printf("Dominant Frequency\nf0=%f\n",f0);
	printf("Maximum Frequency\nfmax=%f\n",fmax);
	printf("Interval of Frequency\ndf=%f\n",df);
	printf("Azimuth of the plane wave divide into the number\nnthita=%d\n",nthita);
	printf("dispersion value\neps=%f\n",eps);
	printf("velocity interval\ndv=%f\n",dv);
	printf("Interval of receiver\ndr=%d\n",dr);
	printf("The number of receivers\nnrec=%d\n",nrec);
	printf("Receiver Z\nr_x=%d\n",r_x);
	printf("LSM-0,TEM-1\niLSTE=%d\n",iLSTE);
	printf("hz=%f\n",hz);
	printf("iNorm=%d\n",iNorm);
	printf("iCompen=%d\n",iCompen);
	printf("Nsmooth=%d\n",Nsmooth);
	printf("wthite_phase=%f\n",wthite_phase);
	printf("whitecoe=%f\n",whitecoe);
	printf("angle=%f\n",angle);
	
	FW2=(float *)malloc(size);
	FW1=(float *)malloc(size);
	FW0=(float *)malloc(size);
	FWb=(float *)malloc(size);
	BW2=(float *)malloc(size);
	BW1=(float *)malloc(size);
	BW0=(float *)malloc(size);
	v=(float *)malloc(size);
	v_2=(float *)malloc(size);
	r=(float *)malloc(size);
	r_1=(float *)malloc(size);
	r_2=(float *)malloc(size);
	MIG1=(float *)malloc(size);
	MIG2=(float *)malloc(size);
	MIG3=(float *)malloc(size);
	Cseis=(float *)malloc(sizeseis);
	seis1=(float **)malloc(sizeof(float *)*(n));
	seis=(float **)malloc(sizeof(float *)*(n));
	for(i=0;i<n;i++)
	{
		seis1[i]=(float *)malloc(sizeof(float )*(NT1));
		seis[i]=(float *)malloc(sizeof(float )*(NT));
	}
	w=(float *)malloc(sizeof(float )*(N2+1));
	for (i=0;i<=N2;i++)
	{
		w[i]=(1.0*i)/(1.0*N2);
	}

	INRE=(float *)malloc(sizeof(float)*(nrec));
	
	DepRec=fopen(OutNameDPR,"r");
	for(i=0;i<nrec;i++)
	{
		fscanf(DepRec,"%f ",&INRE[i]);
	}
	fclose(DepRec);

	velocity(OutNameVp,v,v_2,r,r_1,r_2,NZ,NX,NXZ,N2,tao,h,ifv);

	vmin=v[0*NX+0];
	vmax=v[0*NX+0];

	for(i=0;i<NZ;i++)
	{
		for (j=0;j<NX;j++)
		{
			if(v[i*NX+j]<vmin) vmin=v[i*NX+j];
			if(v[i*NX+j]>vmax) vmax=v[i*NX+j];
		}
	}
	vel=(int(vmin/dv))*dv;
	if (vel>vmin) vmin=vel-dv;
	else vmin=vel;
	vel=(int(vmax/dv))*dv;
	if (vel<vmax) vmax=vel+dv;
	else vmax=vel;
	nvel=int((vmax-vmin)/dv+1.5);

	M=(int *)malloc(sizeof(int)*(nvel));
	fdcoeneed=(int *)malloc(sizeof(int)*(nvel-1));
	Index=(int *)malloc(sizeof(int)*(nvel+1));

	for (i=0;i<nvel;i++)
	{
		fdcoeneed[i]=0;
	}
	for(i=0;i<NZ;i++)
	{
		for(j=0;j<NX;j++)
		{
			k=int((v[i*NX+j]-vmin)/dv+0.5);
			fdcoeneed[k]=1;
		}
	}

	printf("vmin=%f\n",vmin);
	printf("vmax=%f\n",vmax);
	printf("nvel=%d\n",nvel);
	
	if(iLSTE==0)
	{
		NC=funMandC(nthita,nfdmax,nfdmin,nvel,tao,h,df,eps,fmax,vmin,vmax,dv,fdcoeneed,M,Index,iLSTE,&c,hzx);
	}
	else
	{
		NC=nfdmax+1;
		c=(float *)malloc(sizeof(float)*(NC));
		order(2*nfdmax,c);
	}
	
	sizec=sizeof (float) *(NC);
	sizein=sizeof(int)*(nvel+1);
	
	hipMalloc (&DFWADD, size);
	hipMalloc (&DFW2, size);
	hipMalloc (&DFW1, size);
	hipMalloc (&DFW0, size);
	hipMalloc (&DFWb, size);
	hipMalloc (&DBWADD, size);
	hipMalloc (&DBW2, size);
	hipMalloc (&DBW1, size);
	hipMalloc (&DBW0, size);
	hipMalloc (&Dv, size);
	hipMalloc (&Dr, size);
	hipMalloc (&Dr_1, size);
	hipMalloc (&Dw, sizew);
	hipMalloc (&Dc, sizec);
	hipMalloc (&Drel1, size);
	hipMalloc (&Drel2, size);
	hipMalloc (&Dseis, sizeseis);
	hipMalloc (&DIndex, sizein);
	hipMalloc (&Dlf, sizeLF);
	hipMalloc (&Drt, sizeLF);
	hipMalloc (&Dup, sizeUP);
	hipMalloc (&Ddw, sizeUP);

	hipMemcpy (Dv, v, size, hipMemcpyHostToDevice);
	hipMemcpy (Dr, r, size, hipMemcpyHostToDevice);
	hipMemcpy (Dr_1, r_1, size, hipMemcpyHostToDevice);
	hipMemcpy (Dw, w, sizew, hipMemcpyHostToDevice);
	hipMemcpy (Dc, c, sizec, hipMemcpyHostToDevice);
	hipMemcpy (DIndex,Index, sizein, hipMemcpyHostToDevice);
	
	dim3 blockDim (BLOCKSIZE, BLOCKSIZE);
	dim3 gridDim ((NZ+blockDim.x-1)/blockDim.x,(NX+blockDim.y-1)/blockDim.y);
	
	for(m=0;m<nrec;m++)
	{
		printf("/********************the number of %d receiver***********************/\n",m+1);
		N=int(INRE[m]);
		r_u=abs(N/hz)+N2-1;
		printf("r_u=%d r_x=%d N=%d\n",r_u,r_x,N);
	
		for(i=0;i<NXZ;i++)
		{
			FW0[i]=0.0; 
			FW1[i]=0.0;
		}
		FW1[r_u*NX+r_x]=f(0.0,f0)/2.0;

		hipMemcpy (DFW0, FW0, size, hipMemcpyHostToDevice);
		hipMemcpy (DFW1, FW1, size, hipMemcpyHostToDevice);

		Equal <<< gridDim, blockDim >>> (Dlf,Drt,Dup,Ddw,DFW0,DFW1,NZ,NX,N2,nfdmax);
		
		for(k=2;k<NT;k++)
		{
			if(k<NT2) wavelet=f((k-1)*tao,f0);
			else wavelet=0.0;
			if(iLSTE==0) Add <<< gridDim, blockDim >>> (Dv,DFW2,DFW0,DFW1,Dc,DIndex,tao2,h2,vmin,dv,nfdmax,NZ,NX,NXZ,r_u,r_x,wavelet,hzx2_1);
			else
			Add_Con<<< gridDim, blockDim >>> (Dv,DFW2,DFW0,DFW1,Dc,tao2,h2,nfdmax,NZ,NX,NXZ,r_u,r_x,wavelet,hzx2_1);
			Hybrid1 <<< gridDim, blockDim >>> (DFW1,DFW0,DFW2,DFWb,Dlf,Drt,Dup,Ddw,Dv,Dr_1,Dw,NZ,NX,N2,nfdmax,k,taoh,taoh2);
			Hybrid2 <<< gridDim, blockDim >>> (DFW1,DFW0,DFW2,DFWb,Dlf,Drt,Dup,Ddw,Dv,Dr_1,Dw,NZ,NX,N2,nfdmax,k,taoh,taoh2);
			Hybrid3 <<< gridDim, blockDim >>> (DFW1,DFW0,DFW2,DFWb,Dlf,Drt,Dup,Ddw,Dv,Dr_1,Dw,NZ,NX,N2,nfdmax,k,taoh,taoh2);
			Deliver <<< gridDim, blockDim >>> (DFW1,DFW0,DFW2,NZ,NX);
		}
		hipMemcpy (BW0, DFW1, size, hipMemcpyDeviceToHost);
		hipMemcpy (BW1, DFW0, size, hipMemcpyDeviceToHost);
		hipMemcpy (DBW1, BW1, size, hipMemcpyHostToDevice);
		hipMemcpy (DBW0, BW0, size, hipMemcpyHostToDevice);
	
		sprintf(name1,"NEW_L10-1932-X_%d.dat",N);
		strcpy(name3,OutNameseis);
		Rdseismic=fopen(strcat(name3,name1),"rb");
	
		for(i=0;i<n;i++)
		{
			for(k=0;k<NT1;k++)
			{
				fread(&seis1[i][k],sizeof(float),1,Rdseismic);
			}
		}
		fclose(Rdseismic);
		if(NT!=NT1)
		{
			for(i=0;i<n;i++)
			{
				resample (NT1, tao1, seis1[i], NT,tao,seis[i]);
			}		
		}
		else
		{
			for(i=0;i<n;i++)
			{
				for(k=0;k<NT;k++)
				{
					seis[i][k]=seis1[i][k];
				}
			
			}
		}
		printf("NT2=%d,NT1=%d,NT=%d,tao1=%f,tao=%f\n",NT2,NT1,NT,tao1,tao);
		
		for(i=0;i<NXZ;i++)
		{
			FWb[i]=0.0;
			if(iCompen==1)
			{
				BW2[i]=BW0[i]+BW1[i];
				FW2[i]=FW0[i]+FW1[i];
				MIG1[i]=FW2[i]*BW2[i]+FW0[i]*BW0[i];
				MIG2[i]=BW1[i]*BW1[i]+BW0[i]*BW0[i];
			}
			else
			{
				BW2[i]=0.0;	
				FW2[i]=0.0;	
				MIG1[i]=FW1[i]*BW1[i]+FW0[i]*BW0[i];
				MIG2[i]=BW1[i]*BW1[i]+BW0[i]*BW0[i];
			}	
		}
	
		hipMemcpy (DFW2, FW2, size, hipMemcpyHostToDevice);
		hipMemcpy (DFWb, FWb, size, hipMemcpyHostToDevice);
		hipMemcpy (DFW0, FWb, size, hipMemcpyHostToDevice);
		hipMemcpy (DFW1, FWb, size, hipMemcpyHostToDevice);
		hipMemcpy (DFWADD, FW2, size, hipMemcpyHostToDevice);
		hipMemcpy (DBWADD, BW2, size, hipMemcpyHostToDevice);
		hipMemcpy (Drel1,MIG1, size, hipMemcpyHostToDevice);
		hipMemcpy (Drel2,MIG2, size, hipMemcpyHostToDevice);
	
		for(k=NT-3;k>=0;k--)
		{
			if(k<NT2) wavelet=f((k+1)*tao,f0);
			else wavelet=0.0;
			
			BKEqual<<< gridDim, blockDim >>>(Dlf,Drt,Dup,Ddw,DBW1,NZ,NX,N2,nfdmax,k);
			
			if(iLSTE==0)
			{
				BKAdd_EFF <<< gridDim, blockDim >>> (Dv,DBW2,DBW0,DBW1,Dc,DIndex,tao2,h2,vmin,dv,N2,NZ,NX,NXZ,r_u,r_x,wavelet,hzx2_1);
			}
			else
			{
				BKAdd_EFF_Con<<< gridDim, blockDim >>> (Dv,DBW2,DBW0,DBW1,Dc,tao2,h2,N2,NZ,NX,NXZ,r_u,r_x,nfdmax,wavelet,hzx2_1);
			}
			
			Deliver_EFF <<< gridDim, blockDim >>> (DBW1,DBW0,DBW2,N2,NZ,NX);
			
			for(j=0;j<n;j++)
			{
				Cseis[j]=seis[j][k+1];
			}
			hipMemcpy (Dseis, Cseis, sizeseis, hipMemcpyHostToDevice);
			
			if(iLSTE==0)
			{
				BKAdd <<< gridDim, blockDim >>> (Dv,DFW2,DFW1,DFW0,Dc,DIndex,Dseis,tao2,h2,vmin,dv,nfdmax,NZ,NX,NXZ,s_l,s_r,s_z,ds,hzx2_1);
			}
			else
			{
				BKAdd_Con <<< gridDim, blockDim >>> (Dv,DFW2,DFW1,DFW0,Dc,Dseis,tao2,h2,nfdmax,NZ,NX,NXZ,s_l,s_r,s_z,ds,hzx2_1);
			}
			BKHybrid1 <<< gridDim, blockDim >>> (DFW1,DFW0,DFW2,DFWb,Dv,Dr_1,Dw,NZ,NX,N2,taoh,taoh2);
			BKHybrid2 <<< gridDim, blockDim >>> (DFW1,DFW0,DFW2,DFWb,Dv,Dr_1,Dw,NZ,NX,N2,taoh,taoh2);
			Deliver <<< gridDim, blockDim >>> (DFW1,DFW0,DFW2,NZ,NX);
		
			if(iCompen==1)
			{ 
				Rel_Compen <<< gridDim, blockDim >>>(DFW2,DBW2,DFWADD,DBWADD,Drel1,Drel2,NZ,NX); 
			}
			else
			{ 
				Rel_NonCompen <<< gridDim, blockDim >>>(DFW2,DBW2,Drel1,Drel2,NZ,NX);
			}
		}
		hipMemcpy (MIG1, Drel1, size, hipMemcpyDeviceToHost);
		hipMemcpy (MIG2, Drel2, size, hipMemcpyDeviceToHost);
		
		for(j=N2;j<NX-N2;j++)
		{
			for(i=N2;i<NZ-N2;i++)
			{		
				i1=i-1;
				i2=i+1;
				j1=j-1;
				j2=j+1;
				if(i1<N2) i1=2*N2-i1;
				if(j1<N2) j1=2*N2-j1;
				if(i2>=NZ-N2) i2=2*(NZ-N2-1)-i2;
				if(j2>=NX-N2) j2=2*(NX-N2-1)-j2;		
				MIG3[i*NX+j]=-1.0*(MIG1[i*NX+j2]+MIG1[i*NX+j1]+MIG1[i2*NX+j]+MIG1[i1*NX+j]-4*MIG1[i*NX+j])*v[(i)*NX+(j)]*v[(i)*NX+(j)]/(vmax*vmax);						
			}
		}
		
		sprintf(name1,"RVSP_RTM_up_%d.dat",m+1);
		strcpy(name3,Result);
		Af_la=fopen(strcat(name3,name1),"wb");	
		for(j=N2;j<NX-N2;j++)
		{
			for(i=N2;i<NZ-N2;i++)
			{
				fwrite(&MIG3[i*NX+j],sizeof(float),1,Af_la);
			}
		}
		fclose(Af_la);
		
		MIGmax=0.0;
		for(j=N2;j<NX-N2;j++)
		{
			for(i=N2;i<NZ-N2;i++)
			{	
				if(fabs(MIG2[i*NX+j])>MIGmax) MIGmax=fabs(MIG2[i*NX+j]);
			}
		}
		stable=MIGmax*whitecoe;
		printf("%0.16f\n",stable);
		for(j=N2;j<NX-N2;j++)
		{
			for(i=N2;i<NZ-N2;i++)
			{
				MIG2[i*NX+j]=MIG2[i*NX+j]+stable;
			}
		}
		sprintf(name1,"RVSP_RTM_down_%d.dat",m+1);
		strcpy(name3,Result);
		Af_la=fopen(strcat(name3,name1),"wb");	
		for(j=N2;j<NX-N2;j++)
		{
			for(i=N2;i<NZ-N2;i++)
			{
				fwrite(&MIG2[i*NX+j],sizeof(float),1,Af_la);
			}
		}
		fclose(Af_la);
	}
	for(j=0;j<NX;j++)
	{
		for(i=0;i<NZ;i++)
		{							
			FW0[i*NX+j]=0;
			FW1[i*NX+j]=0;
		}
	}
	for(m=0;m<nrec;m++)
	{
		sprintf(name1,"RVSP_RTM_up_%d.dat",m+1);
		strcpy(name3,Result);
		out=fopen(strcat(name3,name1),"rb");
		for(j=0;j<mod_NX;j++)
		{
			for(i=0;i<mod_NZ;i++)
			{
				fread(&MIG1[i*NX+j],sizeof(float),1,out);
			}
		}
		fclose(out);
		
		sprintf(name1,"RVSP_RTM_down_%d.dat",m+1);
		strcpy(name3,Result);
		out=fopen(strcat(name3,name1),"rb");
		for(j=0;j<mod_NX;j++)
		{
			for(i=0;i<mod_NZ;i++)
			{
				fread(&MIG2[i*NX+j],sizeof(float),1,out);
			}
		}
		fclose(out);

		for(j=0;j<mod_NX;j++)
		{
			for(i=0;i<mod_NZ;i++)
			{							
				FW0[i*NX+j]+=MIG1[i*NX+j];				
			}
		}
		for(j=0;j<mod_NX;j++)
		{
			for(i=0;i<mod_NZ;i++)
			{							
				FW1[i*NX+j]+=MIG2[i*NX+j];			
			}
		}
	}

	for(j=0;j<mod_NX;j++)
	{
		for(i=0;i<mod_NZ;i++)
		{
			MIG1[i*NX+j]=FW0[i*NX+j]/nrec;
			MIG2[i*NX+j]=FW1[i*NX+j]/nrec;
		}
	}
	if(iNorm==1)
	{
		for(j=0;j<mod_NX;j++)
		{
			for(i=0;i<mod_NZ;i++)
			{
				MIG1[i*NX+j]=MIG1[i*NX+j]/MIG2[i*NX+j];
			}
		}
	}

	if(ifv==1)
	{
		strcpy(name3,Result);
		out1=fopen(strcat(name3,"RVSP_Migration_Real_new2.dat"),"wb");
		for(j=NX_ED;j>NX_BG;j--)
		{
			for(i=NZ_BG;i<NZ_ED;i++)
			{
				fwrite(&MIG1[i*NX+j],sizeof(float),1,out1);
			}
		}
		fclose(out1);
		
		strcpy(name3,Result);
		out1=fopen(strcat(name3,"vnew.dat"),"wb");
		for(j=NX_ED+N2;j>NX_BG+N2;j--)
		{
			for(i=NZ_BG+N2;i<NZ_ED+N2;i++)
			{
				fwrite(&v[i*NX+j],sizeof(float),1,out1);
			}
		}
		fclose(out1);
	}
	else
	{
		strcpy(name3,Result);
		out1=fopen(strcat(name3,"RVSP_Migration_Real_new2.dat"),"wb");
		for(j=NX_BG;j<NX_ED;j++)
		{
			for(i=NZ_BG;i<NZ_ED;i++)
			{
				fwrite(&MIG1[i*NX+j],sizeof(float),1,out1);
			}
		}
		fclose(out1);
		
		strcpy(name3,Result);
		out1=fopen(strcat(name3,"vnew.dat"),"wb");
		for(j=NX_BG+N2;j<NX_ED+N2;j++)
		{
			for(i=NZ_BG;i<NZ_ED+N2;i++)
			{
				fwrite(&v[i*NX+j],sizeof(float),1,out1);
			}
		}
		fclose(out1);
	}
		
	int SGY_N;
	float *SGYseis,*DSR,*SX,*SY,RX,RY;
	char namesgy[100];
	SGY_N=NX_ED-NX_BG;
	mod_NX=SGY_N;
	float *MIG,*vDT,*MIG1_T,*MIG_T;
	int nt;

	MIG=(float *)malloc(sizeof (float) *(mod_NX*mod_NZ));
	vDT=(float *)malloc(sizeof (float) *(mod_NX*mod_NZ));
	
	
	strcpy(name3,Result);
	out1=fopen(strcat(name3,"RVSP_Migration_Real_new2.dat"),"rb");
	for(i=0;i<mod_NX;i++)
	{
		for(j=0;j<mod_NZ;j++)
		{		
			fread(&MIG[i*mod_NZ+j],sizeof(float),1,out1);
		}
	}
	fclose(out1);
	
	strcpy(name3,Result);
	out1=fopen(strcat(name3,"vnew.dat"),"rb");
	for(i=0;i<mod_NX;i++)
	{
		for(j=0;j<mod_NZ;j++)
		{		
			fread(&vDT[i*mod_NZ+j],sizeof(float),1,out1);
		}
	}
	fclose(out1);
		
	
	sprintf(name1,"RVSP_Migration_Real_T.dat",N);
	strcpy(name3,Result);
	nt=D2T(strcat(name3,name1), vDT, MIG, mod_NX, mod_NZ, 0, (mod_NX-1), hz, hz, tao);
	printf("nt=%d\n",nt);
	MIG_T=(float *)malloc(sizeof (float) *(mod_NX*nt));
	MIG1_T=(float *)malloc(sizeof (float) *(mod_NX*nt));
	
	strcpy(name3,Result);
	out1=fopen(strcat(name3,"RVSP_Migration_Real_T.dat"),"rb");
	for(i=0;i<mod_NX;i++)
	{
		for(j=0;j<nt;j++)
		{		
			fread(&MIG1_T[i*nt+j],sizeof(float),1,out1);
		}
	}
	fclose(out1);

	phase_correction(MIG1_T,MIG_T,mod_NX, nt,angle);
	
	strcpy(name3,Result);
	out1=fopen(strcat(name3,"RVSP_Migration_Real_T_phase.dat"),"wb");
	for(i=0;i<mod_NX;i++)
	{
		for(j=0;j<nt;j++)
		{		
			fwrite(&MIG_T[i*nt+j],sizeof(float),1,out1);
		}
	}
	fclose(out1);
	
	sprintf(name1,"RVSP_Migration_Real_D.dat",N);
	strcpy(name3,Result);
	nt=T2D(strcat(name3,name1),vDT, MIG_T, mod_NX, nt, mod_NZ, 0, (mod_NX-1), hz, tao,hz);
	printf("nt=%d\n",nt);
	
	SGYseis=(float *)malloc(sizeof(float )*(SGY_N*mod_NZ));

	SX=(float *)malloc(sizeof(float )*(SGY_N));
	SY=(float *)malloc(sizeof(float )*(SGY_N));
	DSR=(float *)malloc(sizeof(float )*(SGY_N));	

	strcpy(name3,Result);
	out1=fopen(strcat(name3,"RVSP_Migration_Real_new2.dat"),"rb");
	for(i=0;i<SGY_N;i++)
	{
		for(k=0;k<mod_NZ;k++)
		{
			fread(&SGYseis[i*mod_NZ+k],sizeof(float),1,out1);
		}
	}
	fclose(out1);
	for(i=0;i<SGY_N;i++)
	{
		SX[i]=1.0*i*10;
		SY[i]=-1.0*i*10;
		DSR[i]=1000-i;
	}
	RX=1.0;
	RY=1.0;
	
	strcpy(namesgy,Result);
	strcat(namesgy,"RVSP_migration_Real.sgy");

	WriteSGY(SGYseis,SGY_N,mod_NZ,(int)(hz),SX,SY,RX,RY,DSR,namesgy);
	
	finish = clock ();
	duration = (double) (finish - start) / CLOCKS_PER_SEC;
	printf ("%f seconds\n", duration);

	free(INRE);
	free(w);

	for(i=0;i<n;i++)
	{
		free(seis[i]);
		free(seis1[i]);
	}
	free(seis);
	free(Cseis);
	free(MIG2);
	free(MIG1);
	free(c);
	free(r_2);
	free(r_1);
	free(r);
	free(v_2);
	free(v);
	free(BW0);
	free(BW1);
	free(BW2);
	free(FWb);
	free(FW0);
	free(FW1);
	free(FW2);

	hipFree(DIndex);
	hipFree(Dseis);
	hipFree(Drel2);
	hipFree(Drel1);
	hipFree(Dc);
	hipFree(Dw);
	hipFree(Dr_1);
	hipFree(Dr);
	hipFree(Dv);
	hipFree(DBW0);
	hipFree(DBW1);
	hipFree(DBW2);
	hipFree(DFWb);
	hipFree(DFW0);
	hipFree(DFW1);
	hipFree(DFW2);
}


/**************************�Ӳ�**************************/
float f(float t1,float f0)
{
	float t00=1/f0,y;
	y=(1-2*pow(pi*f0*(t1-t00),2))*exp(-pow(pi*f0*(t1-t00),2)); 
	return(y);
}
